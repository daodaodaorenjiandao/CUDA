#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


int main()
{
	float * pDeviceData = nullptr;
	int width = 10 * sizeof(float);
	int height = 10 * sizeof(float);
	size_t pitch;

	hipError_t err = hipSuccess;

	//1 use cudaMallocPitch function
	err = hipMallocPitch(&pDeviceData, &pitch, width, height);		//ע�������width��height�ĵ�λΪ�ֽ���
	if (err != hipSuccess)
	{
		cout << "call hipMallocPitch fail!!!" << endl;
		exit(1);
	}
	cout << "width: " << width << endl;
	cout << "height: " << height << endl;
	cout << "pitch: " << pitch << endl;

	

	//2 use cudaMalloc3D
	hipPitchedPtr pitchPtr;
	hipExtent extent;
	extent.width = 10 * sizeof(float);
	extent.height = 22 * sizeof(float);
	extent.depth = 33 * sizeof(float);

	err = hipMalloc3D(&pitchPtr, extent);
	if (err != hipSuccess)
	{
		cout << "call hipMalloc3D fail!!!" << endl;
		exit(1);
	}
	cout << "\n\n";
	cout << "width: " << extent.width << endl;			//��������ڴ�ĳ�ʼֵ
	cout << "height: " << extent.height << endl;
	cout << "depth: " << extent.depth << endl;

	cout << endl;
	cout << "pitch: " << pitchPtr.pitch << endl;		//���ʵ�ʵĿ��ֵ
	cout << "xsize: " << pitchPtr.xsize << endl;		//��Ч���--����extent.width
	cout << "ysize: " << pitchPtr.ysize << endl;		//��Ч�߶�--����extent.height

	hipFree(pDeviceData);
	hipFree(pitchPtr.ptr);
	cin.get();
	return 0;
}

