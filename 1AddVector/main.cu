#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//�ú�������Ϊ��__global__����ʾ��GPU����ִ��.
//�书��Ϊ������pA��pB�ж�Ӧλ�õ�������ӣ����������������pC�Ķ�Ӧλ����
//ÿ�������������СΪsize
__global__
void add(const float * pA, const float * pB, float * pC, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;		//���㵱ǰ�����е�����
	if (index < size)										//ȷ����һ����Ч������
		pC[index] = pA[index] + pB[index];

}

int main()
{
	unsigned int numElement = 30000000;
	int totalSize = sizeof(float)* numElement;

	//init
	float *pA = (float*)malloc(totalSize);
	float *pB = (float*)malloc(totalSize);
	float *pC = (float*)malloc(totalSize);

	for (int i = 0; i < numElement; ++i)
	{
		*(pA + i) = rand() / (float)RAND_MAX;;
		*(pB + i) = rand() / (float)RAND_MAX;
	}

	//cpu segment

	//begin use cpu comput
	clock_t startTime, endTime;
	startTime = clock();
	for (int i = 0; i < numElement; ++i)
	{
		*(pC + i) = *(pA + i) + *(pB + i);
	}
	endTime = clock();
	//end use cpu comput

	printf("use cpu comput finish!\n");
	printf("use total time = %fs\n", (endTime - startTime) / 1000.f);
	printf("\n\n");


	//gpu segment
	float *pD, *pE, *pF;
	hipError_t err = hipSuccess;

	//malloc memory
	err = hipMalloc(&pD, totalSize);
	if (err != hipSuccess)
	{
		printf("call hipMalloc fail for pD.\n");
		exit(1);
	}

	err = hipMalloc(&pE, totalSize);
	if (err != hipSuccess)
	{
		printf("call hipMalloc fail for pE.\n");
		exit(1);
	}

	err = hipMalloc(&pF, totalSize);
	if (err != hipSuccess)
	{
		printf("call hipMalloc fail for pF.\n");
		exit(1);
	}

	//copy data  from pA pB pC to pD pE pF
	err = hipMemcpy(pD, pA, totalSize, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("call hipMemcpy fail for pA to pD.\n");
		exit(1);
	}

	err = hipMemcpy(pE, pB, totalSize, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("call hipMemcpy fail for pB to pE.\n");
		exit(1);
	}


	//begin use gpu comput
	startTime = clock();
	int threadPerBlock = 1024;
	int numBlock = (numElement - 1) / threadPerBlock + 1;
	add << <numBlock, threadPerBlock >> >(pD, pE, pF, numElement);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("use gpu comput fail!\n");
		exit(1); 
	}

	endTime = clock();
	printf("use gpu comput finish!\n");
	printf("use time : %fs\n",(endTime - startTime) / 1000.f);
	//end use gpu comput


	//copu data from device to host
	err = hipMemcpy(pC, pF, numElement, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("call hipMemcpy form pF to pC fail.\n");
		exit(1);
	}

	//check data
	for (int i = 0; i < numElement; ++i)
	{
		if (fabs(pA[i] + pB[i] - pC[i]) > 1e-5)
		{
			printf("%f + %f != %f\n",pA[i],pB[i],pC[i]);
		}
	}

	//�ͷ��豸�ϵ��ڴ�
	hipFree(pD);
	hipFree(pE);
	hipFree(pF);

	//�ڳ����˳�ǰ�����øú������ø��豸��ʹ����ȥ�����豸״̬�������ڳ����˳�ǰ���е����ݽ���ˢ����
	err = hipDeviceReset();
	if (err != hipSuccess)
	{
		printf("call hipDeviceReset fail!\n");
		exit(1);
	}

	free(pA);
	free(pB);
	free(pC);

	getchar();
	return 0;
}