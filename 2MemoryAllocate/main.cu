#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


int main()
{
	float * pDeviceData = nullptr;
	int width = 10 * sizeof(float);
	int height = 10 * sizeof(float);
	size_t pitch;

	hipError_t err = hipSuccess;

	//1 use cudaMallocPitch function
	err = hipMallocPitch(&pDeviceData, &pitch, width, height);		//注意这里的width和height的单位为字节数
	if (err != hipSuccess)
	{
		cout << "call hipMallocPitch fail!!!" << endl;
		exit(1);
	}
	cout << "width: " << width << endl;
	cout << "height: " << height << endl;
	cout << "pitch: " << pitch << endl;

	

	//2 use cudaMalloc3D
	hipPitchedPtr pitchPtr;
	hipExtent extent;
	extent.width = 10 * sizeof(float);
	extent.height = 22 * sizeof(float);
	extent.depth = 33 * sizeof(float);

	err = hipMalloc3D(&pitchPtr, extent);
	if (err != hipSuccess)
	{
		cout << "call hipMalloc3D fail!!!" << endl;
		exit(1);
	}
	cout << "\n\n";
	cout << "width: " << extent.width << endl;			//输出申请内存的初始值
	cout << "height: " << extent.height << endl;
	cout << "depth: " << extent.depth << endl;

	cout << endl;
	cout << "pitch: " << pitchPtr.pitch << endl;		//输出实际的宽度值
	cout << "xsize: " << pitchPtr.xsize << endl;		//有效宽度--等于extent.width
	cout << "ysize: " << pitchPtr.ysize << endl;		//有效高度--等于extent.height

	hipFree(pDeviceData);
	hipFree(pitchPtr.ptr);
	cin.get();
	return 0;
}

